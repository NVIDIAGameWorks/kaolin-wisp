#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION & AFFILIATES.  All rights reserved.
 *
 * NVIDIA CORPORATION & AFFILIATES and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION & AFFILIATES is strictly prohibited.
 */

#include <ATen/ATen.h>
#include <c10/cuda/CUDAGuard.h>

namespace wisp {
typedef unsigned int uint;

__device__ int32_t 
hash_index(
    const int3 pos,
    const int32_t resolution,
    const int32_t codebook_size
){
    int32_t index = 0;

    constexpr uint32_t primes[3] = { 1u, 2654435761u, 805459861u };

    if (resolution < codebook_size && 
        resolution * resolution < codebook_size && 
        resolution * resolution * resolution < codebook_size) {
        index = pos.x + 
                pos.y * resolution + 
                pos.z * resolution * resolution;
    } else {
        index = (pos.x * primes[0] ^
                 pos.y * primes[1] ^
                 pos.z * primes[2]) % codebook_size;
    }
    return index;
}

__inline__ __device__ float 
clamp(float x, float a, float b)
{
    return max(a, min(b, x));
}

template<typename scalar_t>
__global__ void
hashgrid_interpolate_cuda_kernel(
    const int64_t num_coords,
    const int32_t codebook_size,
    const int64_t feature_dim,
    const int32_t resolution,
    const int32_t lod_idx,
    const int32_t num_lods,
    const float* __restrict__ coords,
    const scalar_t* __restrict__ codebook,
    scalar_t* __restrict__ feats
){
    uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int64_t stride = blockDim.x*gridDim.x;
    for (int64_t i=tidx; i<num_coords; i+=stride) { 
        
        float3 x = make_float3(clamp(resolution * (coords[i*3+0] * 0.5 + 0.5), 0, resolution-1-1e-5), 
                               clamp(resolution * (coords[i*3+1] * 0.5 + 0.5), 0, resolution-1-1e-5), 
                               clamp(resolution * (coords[i*3+2] * 0.5 + 0.5), 0, resolution-1-1e-5));
        int3 pos = make_int3(floor(x.x), floor(x.y), floor(x.z));
        float3 x_ = make_float3(x.x - static_cast<float>(pos.x), 
                                x.y - static_cast<float>(pos.y), 
                                x.z - static_cast<float>(pos.z));
        float3 _x = make_float3(1.0 - x_.x, 1.0 - x_.y, 1.0 - x_.z);

        float c000 = _x.x * _x.y * _x.z;
        float c001 = _x.x * _x.y * x_.z;
        float c010 = _x.x * x_.y * _x.z;
        float c011 = _x.x * x_.y * x_.z;
        float c100 = x_.x * _x.y * _x.z;
        float c101 = x_.x * _x.y * x_.z;
        float c110 = x_.x * x_.y * _x.z;
        float c111 = x_.x * x_.y * x_.z;
        
        int32_t corner_idx[8];
#       pragma unroll
        for (int j=0; j<8; ++j) {
            int3 corner;
            corner.x = pos.x + ((j & 4) >> 2);
            corner.y = pos.y + ((j & 2) >> 1);
            corner.z = pos.z + ((j & 1) >> 0);
            corner_idx[j] = hash_index(corner, resolution, codebook_size);
        }
        
        for (uint64_t j=0; j<feature_dim; ++j) {
            float feat =
                static_cast<float>(codebook[corner_idx[0]*feature_dim+j]) * c000 + 
                static_cast<float>(codebook[corner_idx[1]*feature_dim+j]) * c001 + 
                static_cast<float>(codebook[corner_idx[2]*feature_dim+j]) * c010 + 
                static_cast<float>(codebook[corner_idx[3]*feature_dim+j]) * c011 +
                static_cast<float>(codebook[corner_idx[4]*feature_dim+j]) * c100 + 
                static_cast<float>(codebook[corner_idx[5]*feature_dim+j]) * c101 + 
                static_cast<float>(codebook[corner_idx[6]*feature_dim+j]) * c110 +
                static_cast<float>(codebook[corner_idx[7]*feature_dim+j]) * c111;
            feats[num_lods*i*feature_dim+feature_dim*lod_idx+j] = static_cast<scalar_t>(feat);
        }
    }
} 

void hashgrid_interpolate_cuda_impl(
    int64_t num_coords, 
    int32_t codebook_size,
    int64_t feature_dim,
    int32_t resolution,
    int32_t lod_idx,
    int32_t num_lods,
    at::Tensor coords,
    at::Tensor codebook,
    at::Tensor feats){

    int num_threads = 512;
    
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(feats.type(), "hashgrid_interpolate_cuda", ([&] {
        const at::cuda::OptionalCUDAGuard device_guard(at::device_of(feats));
        auto stream = at::cuda::getCurrentCUDAStream();
        hashgrid_interpolate_cuda_kernel<<<(num_coords + num_threads - 1) / num_threads, num_threads, 0, stream>>>(
            num_coords,
            codebook_size,
            feature_dim,
            resolution,
            lod_idx,
            num_lods,
            coords.data_ptr<float>(),
            codebook.data_ptr<scalar_t>(),
            feats.data_ptr<scalar_t>()
        );
    }));
}

template<typename scalar_t>
__global__ void
hashgrid_interpolate_backward_cuda_kernel(
    const int64_t num_coords,
    const int32_t codebook_size,
    const int64_t feature_dim,
    const int32_t resolution,
    const int32_t lod_idx,
    const int32_t num_lods,
    const float* __restrict__ coords,
    const scalar_t* __restrict__ grad_output, // N, feature_dim*num_lods
    scalar_t* __restrict__ grad_codebook // codebook_size, feature_dim
){
    uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int64_t stride = blockDim.x*gridDim.x;
    for (int64_t i=tidx; i<num_coords; i+=stride) { 
        
        float3 x = make_float3(clamp(resolution * (coords[i*3+0] * 0.5 + 0.5), 0, resolution-1-1e-5), 
                               clamp(resolution * (coords[i*3+1] * 0.5 + 0.5), 0, resolution-1-1e-5), 
                               clamp(resolution * (coords[i*3+2] * 0.5 + 0.5), 0, resolution-1-1e-5));
        int3 pos = make_int3(floor(x.x), floor(x.y), floor(x.z));
        float3 x_ = make_float3(x.x - static_cast<float>(pos.x), 
                                x.y - static_cast<float>(pos.y), 
                                x.z - static_cast<float>(pos.z));
        float3 _x = make_float3(1.0 - x_.x, 1.0 - x_.y, 1.0 - x_.z);


        float coeffs[8];
        coeffs[0] = _x.x * _x.y * _x.z;
        coeffs[1] = _x.x * _x.y * x_.z;
        coeffs[2] = _x.x * x_.y * _x.z;
        coeffs[3] = _x.x * x_.y * x_.z;
        coeffs[4] = x_.x * _x.y * _x.z;
        coeffs[5] = x_.x * _x.y * x_.z;
        coeffs[6] = x_.x * x_.y * _x.z;
        coeffs[7] = x_.x * x_.y * x_.z;
        
        int32_t corner_idx[8];

#       pragma unroll
        for (int j=0; j<8; ++j) {
            int3 corner;
            corner.x = pos.x + ((j & 4) >> 2);
            corner.y = pos.y + ((j & 2) >> 1);
            corner.z = pos.z + ((j & 1) >> 0);
            corner_idx[j] = hash_index(corner, resolution, codebook_size);
        }
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 600
        if (std::is_same<scalar_t, at::Half>::value) {
            for (uint64_t j=0; j<feature_dim; j += 2) {
#           pragma unroll
                for (int k=0; k<8; ++k) {
                    uint64_t _idx = i*num_lods*feature_dim + lod_idx*feature_dim;
                    __half2 grad = reinterpret_cast<const __half2*>(grad_output)[(_idx + j) / 2];
                    grad = __floats2half2_rn(__half2float(grad.x) * coeffs[k],
                                             __half2float(grad.y) * coeffs[k]);
                    atomicAdd((__half2*)(grad_codebook + (corner_idx[k]*feature_dim + j)), grad);
                }
            }
        } else
#endif
        {
            for (uint64_t j=0; j<feature_dim; ++j) {
#           pragma unroll
                for (int k=0; k<8; ++k) {
                    float grad =
                        grad_output[i*num_lods*feature_dim + lod_idx*feature_dim + j] * coeffs[k];
                    atomicAdd((float*)(grad_codebook + (corner_idx[k]*feature_dim + j)), grad);
                }
            }
        }
    }
}

void hashgrid_interpolate_backward_cuda_impl(
    int64_t num_coords, 
    int32_t codebook_size,
    int64_t feature_dim,
    int32_t resolution,
    int32_t lod_idx,
    int32_t num_lods,
    at::Tensor coords,
    at::Tensor grad_output,
    at::Tensor grad_codebook){

    int num_threads = 512;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_output.type(), "hashgrid_interpolate_backward_cuda", ([&] {
        const at::cuda::OptionalCUDAGuard device_guard(at::device_of(grad_codebook));
        auto stream = at::cuda::getCurrentCUDAStream();
        hashgrid_interpolate_backward_cuda_kernel<<<(num_coords + num_threads - 1) / num_threads, num_threads, 0, stream>>>(
            num_coords,
            codebook_size,
            feature_dim,
            resolution,
            lod_idx,
            num_lods,
            coords.data_ptr<float>(),
            grad_output.data_ptr<scalar_t>(),
            grad_codebook.data_ptr<scalar_t>()
        );
    }));
}

} // namespace wisp

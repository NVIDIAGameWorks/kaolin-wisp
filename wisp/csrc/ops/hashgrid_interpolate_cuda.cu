#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION & AFFILIATES.  All rights reserved.
 *
 * NVIDIA CORPORATION & AFFILIATES and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION & AFFILIATES is strictly prohibited.
 */

#include <iostream>
#include <ATen/ATen.h>
#include <c10/cuda/CUDAGuard.h>
#include "hash_utils.cuh"

namespace wisp {
typedef unsigned int uint;

template<typename scalar_t>
__global__ void
hashgrid_interpolate_3d_cuda_kernel(
    const int64_t num_coords,
    const int32_t codebook_size,
    const int64_t feature_dim,
    const int32_t resolution,
    const int32_t lod_idx,
    const int32_t num_lods,
    const float* __restrict__ coords,
    const scalar_t* __restrict__ codebook,
    const int64_t *codebook_first_idx,
    scalar_t* __restrict__ feats
){
    uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int64_t stride = blockDim.x*gridDim.x;

    codebook = codebook + codebook_first_idx[lod_idx] * feature_dim; 

    for (int64_t i=tidx; i<num_coords; i+=stride) { 
        
        float3 x = make_float3(clamp(resolution * (coords[i*3+0] * 0.5 + 0.5), 0, resolution-1-1e-5), 
                               clamp(resolution * (coords[i*3+1] * 0.5 + 0.5), 0, resolution-1-1e-5), 
                               clamp(resolution * (coords[i*3+2] * 0.5 + 0.5), 0, resolution-1-1e-5));
        int3 pos = make_int3(floor(x.x), floor(x.y), floor(x.z));
        float3 x_ = make_float3(x.x - static_cast<float>(pos.x), 
                                x.y - static_cast<float>(pos.y), 
                                x.z - static_cast<float>(pos.z));
        float3 _x = make_float3(1.0 - x_.x, 1.0 - x_.y, 1.0 - x_.z);

        float c000 = _x.x * _x.y * _x.z;
        float c001 = _x.x * _x.y * x_.z;
        float c010 = _x.x * x_.y * _x.z;
        float c011 = _x.x * x_.y * x_.z;
        float c100 = x_.x * _x.y * _x.z;
        float c101 = x_.x * _x.y * x_.z;
        float c110 = x_.x * x_.y * _x.z;
        float c111 = x_.x * x_.y * x_.z;
        
        int32_t corner_idx[8];
#       pragma unroll
        for (int j=0; j<8; ++j) {
            int3 corner;
            corner.x = pos.x + ((j & 4) >> 2);
            corner.y = pos.y + ((j & 2) >> 1);
            corner.z = pos.z + ((j & 1) >> 0);
            corner_idx[j] = hash_index_3d(corner, resolution, codebook_size);
        }
        
        for (uint64_t j=0; j<feature_dim; ++j) {
            float feat =
                static_cast<float>(codebook[corner_idx[0]*feature_dim+j]) * c000 + 
                static_cast<float>(codebook[corner_idx[1]*feature_dim+j]) * c001 + 
                static_cast<float>(codebook[corner_idx[2]*feature_dim+j]) * c010 + 
                static_cast<float>(codebook[corner_idx[3]*feature_dim+j]) * c011 +
                static_cast<float>(codebook[corner_idx[4]*feature_dim+j]) * c100 + 
                static_cast<float>(codebook[corner_idx[5]*feature_dim+j]) * c101 + 
                static_cast<float>(codebook[corner_idx[6]*feature_dim+j]) * c110 +
                static_cast<float>(codebook[corner_idx[7]*feature_dim+j]) * c111;
            feats[num_lods*i*feature_dim+feature_dim*lod_idx+j] = static_cast<scalar_t>(feat);
        }
    }
} 

template<typename scalar_t>
__global__ void
hashgrid_interpolate_3d_backward_cuda_kernel(
    const int64_t num_coords,
    const int32_t codebook_size,
    const int64_t feature_dim,
    const int32_t resolution,
    const int32_t lod_idx,
    const int32_t num_lods,
    const bool require_grad_coords,
    const float* __restrict__ coords,
    const scalar_t* __restrict__ codebook,
    const int64_t *__restrict__ codebook_first_idx,
    const scalar_t* __restrict__ grad_output, // N, feature_dim*num_lods
    scalar_t* __restrict__ grad_codebook, // codebook_size, feature_dim
    float* __restrict__ grad_coords // N, 3
){
    uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int64_t stride = blockDim.x*gridDim.x;

    grad_codebook = grad_codebook + codebook_first_idx[lod_idx] * feature_dim;
    codebook = codebook + codebook_first_idx[lod_idx] * feature_dim; 

    for (int64_t i=tidx; i<num_coords; i+=stride) { 
        
        float3 x = make_float3(clamp(resolution * (coords[i*3+0] * 0.5 + 0.5), 0, resolution-1-1e-5), 
                               clamp(resolution * (coords[i*3+1] * 0.5 + 0.5), 0, resolution-1-1e-5), 
                               clamp(resolution * (coords[i*3+2] * 0.5 + 0.5), 0, resolution-1-1e-5));
        int3 pos = make_int3(floor(x.x), floor(x.y), floor(x.z));
        float3 x_ = make_float3(x.x - static_cast<float>(pos.x), 
                                x.y - static_cast<float>(pos.y), 
                                x.z - static_cast<float>(pos.z));
        float3 _x = make_float3(1.0 - x_.x, 1.0 - x_.y, 1.0 - x_.z);


        float coeffs[8];
        coeffs[0] = _x.x * _x.y * _x.z;
        coeffs[1] = _x.x * _x.y * x_.z;
        coeffs[2] = _x.x * x_.y * _x.z;
        coeffs[3] = _x.x * x_.y * x_.z;
        coeffs[4] = x_.x * _x.y * _x.z;
        coeffs[5] = x_.x * _x.y * x_.z;
        coeffs[6] = x_.x * x_.y * _x.z;
        coeffs[7] = x_.x * x_.y * x_.z;
        
        int32_t corner_idx[8];

#       pragma unroll
        for (int j=0; j<8; ++j) {
            int3 corner;
            corner.x = pos.x + ((j & 4) >> 2);
            corner.y = pos.y + ((j & 2) >> 1);
            corner.z = pos.z + ((j & 1) >> 0);
            corner_idx[j] = hash_index_3d(corner, resolution, codebook_size);
        }
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 600
        if (std::is_same<scalar_t, at::Half>::value) {
            for (uint64_t j=0; j<feature_dim; j += 2) {
#           pragma unroll
                for (int k=0; k<8; ++k) {
                    uint64_t _idx = i*num_lods*feature_dim + lod_idx*feature_dim + j;
                    __half2 grad = reinterpret_cast<const __half2*>(grad_output)[_idx / 2];
                    grad = __floats2half2_rn(__half2float(grad.x) * coeffs[k],
                                             __half2float(grad.y) * coeffs[k]);
                    atomicAdd((__half2*)(grad_codebook + (corner_idx[k]*feature_dim + j)), grad);
                }
            }
        } else
#endif
        {
            for (uint64_t j=0; j<feature_dim; ++j) {
#           pragma unroll
                for (int k=0; k<8; ++k) {
                    float grad =
                        grad_output[i*num_lods*feature_dim + lod_idx*feature_dim + j] * coeffs[k];
                    atomicAdd((float*)(grad_codebook + (corner_idx[k]*feature_dim + j)), grad);
                }
            }
        }
        
        if (require_grad_coords) {
            for (uint64_t j=0; j<feature_dim; ++j) {
                // FIX IN MASTER lod_idx
                float _grad_output = static_cast<float>(grad_output[i*num_lods*feature_dim+j]);

                grad_coords[i*3 + 0] += _grad_output * 
                    ((_x.y * _x.z) * 
                    (static_cast<float>(codebook[corner_idx[4]*feature_dim+j]) -
                     static_cast<float>(codebook[corner_idx[0]*feature_dim+j])) +
                    (_x.y * x_.z) * 
                    (static_cast<float>(codebook[corner_idx[5]*feature_dim+j]) -
                     static_cast<float>(codebook[corner_idx[1]*feature_dim+j])) +
                     (x_.y * _x.z) * 
                    (static_cast<float>(codebook[corner_idx[6]*feature_dim+j]) -
                     static_cast<float>(codebook[corner_idx[2]*feature_dim+j])) +
                     (x_.y * x_.z) * 
                    (static_cast<float>(codebook[corner_idx[7]*feature_dim+j]) -
                     static_cast<float>(codebook[corner_idx[3]*feature_dim+j])));
                
                grad_coords[i*3 + 1] += _grad_output * 
                    ((_x.x * _x.z) * 
                    (static_cast<float>(codebook[corner_idx[2]*feature_dim+j]) -
                     static_cast<float>(codebook[corner_idx[0]*feature_dim+j])) +
                    (_x.x * x_.z) * 
                    (static_cast<float>(codebook[corner_idx[3]*feature_dim+j]) -
                     static_cast<float>(codebook[corner_idx[1]*feature_dim+j])) +
                     (x_.x * _x.z) * 
                    (static_cast<float>(codebook[corner_idx[6]*feature_dim+j]) -
                     static_cast<float>(codebook[corner_idx[4]*feature_dim+j])) +
                     (x_.x * x_.z) * 
                    (static_cast<float>(codebook[corner_idx[7]*feature_dim+j]) -
                     static_cast<float>(codebook[corner_idx[6]*feature_dim+j])));
                
                grad_coords[i*3 + 2] += _grad_output * 
                    ((_x.x * _x.y) * 
                    (static_cast<float>(codebook[corner_idx[1]*feature_dim+j]) -
                     static_cast<float>(codebook[corner_idx[0]*feature_dim+j])) +
                    (_x.x * x_.y) * 
                    (static_cast<float>(codebook[corner_idx[3]*feature_dim+j]) -
                     static_cast<float>(codebook[corner_idx[2]*feature_dim+j])) +
                     (x_.x * _x.y) * 
                    (static_cast<float>(codebook[corner_idx[5]*feature_dim+j]) -
                     static_cast<float>(codebook[corner_idx[4]*feature_dim+j])) +
                     (x_.x * x_.y) * 
                    (static_cast<float>(codebook[corner_idx[7]*feature_dim+j]) -
                     static_cast<float>(codebook[corner_idx[6]*feature_dim+j])));
            }   
        }
    }
}

template<typename scalar_t>
__global__ void
hashgrid_interpolate_2d_cuda_kernel(
    const int64_t num_coords,
    const int32_t codebook_size,
    const int64_t feature_dim,
    const int32_t resolution_x,
    const int32_t resolution_y,
    const int32_t lod_idx,
    const int32_t num_lods,
    const float* __restrict__ coords,
    const scalar_t* __restrict__ codebook,
    const int64_t *codebook_first_idx,
    scalar_t* __restrict__ feats
){
    uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int64_t stride = blockDim.x*gridDim.x;
    
    codebook = codebook + codebook_first_idx[lod_idx] * feature_dim; 
    
    for (int64_t i=tidx; i<num_coords; i+=stride) { 
        
        float2 x = make_float2(clamp(resolution_x * (coords[i*2+0] * 0.5 + 0.5), 0, resolution_x-1-1e-5), 
                               clamp(resolution_y * (coords[i*2+1] * 0.5 + 0.5), 0, resolution_y-1-1e-5));
        int2 pos = make_int2(floor(x.x), floor(x.y));
        float2 x_ = make_float2(x.x - static_cast<float>(pos.x), 
                                x.y - static_cast<float>(pos.y));
        float2 _x = make_float2(1.0 - x_.x, 1.0 - x_.y);

        float c00 = _x.x * _x.y;
        float c01 = _x.x * x_.y;
        float c10 = x_.x * _x.y;
        float c11 = x_.x * x_.y;
        
        int32_t corner_idx[4];
#       pragma unroll
        for (uint32_t j=0; j<4; ++j) {
            int2 corner;
            corner.x = pos.x + ((j & 2) >> 1);
            corner.y = pos.y + ((j & 1) >> 0);
            corner_idx[j] = hash_index_2d(corner, resolution_x, resolution_y, codebook_size);
        }
        
        for (uint32_t j=0; j<feature_dim; ++j) {
            float feat =
                static_cast<float>(codebook[corner_idx[0]*feature_dim+j]) * c00 + 
                static_cast<float>(codebook[corner_idx[1]*feature_dim+j]) * c01 + 
                static_cast<float>(codebook[corner_idx[2]*feature_dim+j]) * c10 + 
                static_cast<float>(codebook[corner_idx[3]*feature_dim+j]) * c11;
            feats[num_lods*i*feature_dim+feature_dim*lod_idx+j] = static_cast<scalar_t>(feat);
        }
    }
} 


template<typename scalar_t>
__global__ void
hashgrid_interpolate_2d_backward_cuda_kernel(
    const int64_t num_coords,
    const int32_t codebook_size,
    const int64_t feature_dim,
    const int32_t resolution_x,
    const int32_t resolution_y,
    const int32_t lod_idx,
    const int32_t num_lods,
    const bool require_grad_coords,
    const float* __restrict__ coords,
    const scalar_t* __restrict__ codebook,
    const int64_t *codebook_first_idx,
    const scalar_t* __restrict__ grad_output, // N, feature_dim*num_lods
    scalar_t* __restrict__ grad_codebook, // codebook_size, feature_dim
    float* __restrict__ grad_coords // N, 3
){
    uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int64_t stride = blockDim.x*gridDim.x;
    
    grad_codebook = grad_codebook + codebook_first_idx[lod_idx] * feature_dim;
    codebook = codebook + codebook_first_idx[lod_idx] * feature_dim; 
    
    for (int64_t i=tidx; i<num_coords; i+=stride) { 
        
        float2 x = make_float2(clamp(resolution_x * (coords[i*2+0] * 0.5 + 0.5), 0, resolution_x-1-1e-5), 
                               clamp(resolution_y * (coords[i*2+1] * 0.5 + 0.5), 0, resolution_y-1-1e-5));
        int2 pos = make_int2(floor(x.x), floor(x.y));
        float2 x_ = make_float2(x.x - static_cast<float>(pos.x), 
                                x.y - static_cast<float>(pos.y));
        float2 _x = make_float2(1.0 - x_.x, 1.0 - x_.y);

        float coeffs[4];
        coeffs[0] = _x.x * _x.y;
        coeffs[1] = _x.x * x_.y;
        coeffs[2] = x_.x * _x.y;
        coeffs[3] = x_.x * x_.y;
        
        int32_t corner_idx[4];
#       pragma unroll
        for (uint32_t j=0; j<4; ++j) {
            int2 corner;
            corner.x = pos.x + ((j & 2) >> 1);
            corner.y = pos.y + ((j & 1) >> 0);
            corner_idx[j] = hash_index_2d(corner, resolution_x, resolution_y, codebook_size);
        }
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 600
        if (std::is_same<scalar_t, at::Half>::value) {
            for (uint32_t j=0; j<feature_dim; j += 2) {
#           pragma unroll
                for (uint32_t k=0; k<4; ++k) {
                    uint64_t _idx = i*num_lods*feature_dim + lod_idx*feature_dim + j;
                    __half2 grad = reinterpret_cast<const __half2*>(grad_output)[_idx / 2];
                    grad = __floats2half2_rn(__half2float(grad.x) * coeffs[k],
                                             __half2float(grad.y) * coeffs[k]);
                    atomicAdd((__half2*)(grad_codebook + (corner_idx[k]*feature_dim + j)), grad);
                }
            }
        } else
#endif
        {
            for (uint32_t j=0; j<feature_dim; ++j) {
#           pragma unroll
                for (uint32_t k=0; k<4; ++k) {
                    float grad =
                        grad_output[i*num_lods*feature_dim + lod_idx*feature_dim + j] * coeffs[k];
                    atomicAdd((float*)(grad_codebook + (corner_idx[k]*feature_dim + j)), grad);
                }
            }
        }
    }
}


void hashgrid_interpolate_cuda_impl(
    int64_t num_coords, 
    int32_t codebook_size,
    int64_t feature_dim,
    at::Tensor resolution,
    int32_t lod_idx,
    int32_t num_lods,
    int32_t coord_dim,
    at::Tensor coords,
    at::Tensor codebook,
    at::Tensor codebook_first_idx,
    at::Tensor feats){

    int num_threads = 512;
    
    if (coord_dim == 3) {
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(feats.type(), "hashgrid_interpolate_3d_cuda", ([&] {
            const at::cuda::OptionalCUDAGuard device_guard(at::device_of(feats));
            auto stream = at::cuda::getCurrentCUDAStream();
            hashgrid_interpolate_3d_cuda_kernel<<<(num_coords + num_threads - 1) / num_threads, num_threads, 0, stream>>>(
                num_coords,
                codebook_size,
                feature_dim,
                resolution[0].item<int>(),
                lod_idx,
                num_lods,
                coords.data_ptr<float>(),
                codebook.data_ptr<scalar_t>(),
                codebook_first_idx.data_ptr<int64_t>(),
                feats.data_ptr<scalar_t>()
            );
        }));
    } else if (coord_dim == 2) {
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(feats.type(), "hashgrid_interpolate_2d_cuda", ([&] {
            const at::cuda::OptionalCUDAGuard device_guard(at::device_of(feats));
            auto stream = at::cuda::getCurrentCUDAStream();
            hashgrid_interpolate_2d_cuda_kernel<<<(num_coords + num_threads - 1) / num_threads, num_threads, 0, stream>>>(
                num_coords,
                codebook_size,
                feature_dim,
                resolution[0].item<int>(),
                resolution[1].item<int>(),
                lod_idx,
                num_lods,
                coords.data_ptr<float>(),
                codebook.data_ptr<scalar_t>(),
                codebook_first_idx.data_ptr<int64_t>(),
                feats.data_ptr<scalar_t>()
            );
        }));
    }
}

void hashgrid_interpolate_backward_cuda_impl(
    int64_t num_coords, 
    int32_t codebook_size,
    int64_t feature_dim,
    at::Tensor resolution,
    int32_t lod_idx,
    int32_t num_lods,
    int32_t coord_dim,
    bool require_grad_coords,
    at::Tensor coords,
    at::Tensor codebook,
    at::Tensor codebook_first_idx,
    at::Tensor grad_output,
    at::Tensor grad_codebook,
    at::Tensor grad_coords){

    int num_threads = 512;

    if (coord_dim == 3) {
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_output.type(), "hashgrid_interpolate_3d_backward_cuda", ([&] {
            const at::cuda::OptionalCUDAGuard device_guard(at::device_of(grad_codebook));
            auto stream = at::cuda::getCurrentCUDAStream();
            hashgrid_interpolate_3d_backward_cuda_kernel<<<(num_coords + num_threads - 1) / num_threads, num_threads, 0, stream>>>(
                num_coords,
                codebook_size,
                feature_dim,
                resolution[0].item<int>(),
                lod_idx,
                num_lods,
                require_grad_coords,
                coords.data_ptr<float>(),
                codebook.data_ptr<scalar_t>(),
                codebook_first_idx.data_ptr<int64_t>(),
                grad_output.data_ptr<scalar_t>(),
                grad_codebook.data_ptr<scalar_t>(),
                grad_coords.data_ptr<float>()
            );
        }));
    } else if (coord_dim == 2) {
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_output.type(), "hashgrid_interpolate_2d_backward_cuda", ([&] {
            const at::cuda::OptionalCUDAGuard device_guard(at::device_of(grad_codebook));
            auto stream = at::cuda::getCurrentCUDAStream();
            hashgrid_interpolate_2d_backward_cuda_kernel<<<(num_coords + num_threads - 1) / num_threads, num_threads, 0, stream>>>(
                num_coords,
                codebook_size,
                feature_dim,
                resolution[0].item<int>(),
                resolution[1].item<int>(),
                lod_idx,
                num_lods,
                require_grad_coords,
                coords.data_ptr<float>(),
                codebook.data_ptr<scalar_t>(),
                codebook_first_idx.data_ptr<int64_t>(),
                grad_output.data_ptr<scalar_t>(),
                grad_codebook.data_ptr<scalar_t>(),
                grad_coords.data_ptr<float>()
            );
        }));
    }
}

} // namespace wisp
